#include "hip/hip_runtime.h"
/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       **
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **
*                                                                                          **
* Copyright (c) 2008 International Institute of Information Technology.                    **
* All rights reserved.                                                                     **
*                                                                                          **
* Permission to use, copy, modify and distribute this software and its documentation for   **
* educational purpose is hereby granted without fee, provided that the above copyright     **
* notice and this permission notice appear in all copies of this software and that you do  **
* not sell the software.                                                                   **
*                                                                                          **
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR    **
* OTHERWISE.                                                                               **
*                                                                                          **
* Created By Vibhav Vineet.                                                                **
********************************************************************************************/

#include "CudaCuts.h"

#include <algorithm>

using namespace std;

typedef struct inputfile_t
{
	char* filename;
	int width;
	int height;
	int numOfLabels;
	int* dataterm_error;
	int* smoothness_table;
	int* hcue;
	int* vcue;
}inputfile_t;

void loadFile(inputfile_t& input_file);
void writePGM(const char *filename, CudaCuts& cuts);

int main(int argc, char* argv[])
{
	if(argc != 2)
	{
		printf("usage: <datafilename.txt>");
	}
	char* dataFile = argv[1];

	// Read input file
	inputfile_t input_file;
	input_file.filename = dataFile;
	loadFile(input_file);

	// Set-up graph with input
	CudaCuts cuts(input_file.width, input_file.height, input_file.numOfLabels
				, input_file.dataterm_error, input_file.smoothness_table
				, input_file.hcue, input_file.vcue);

	// Generate input labels, 0 to (numOfLabels)
	std::vector<int> labels(input_file.numOfLabels);
	int n=0;
	generate(labels.begin(), labels.end(), [&n] { return n++;});
	cuts.run(labels);
	
	//save output file
	const char* output_name = "result_sponge/flower_cuda_test.pgm";
	writePGM(output_name, cuts);

	return 0;
}

void writePGM(const char* filename, CudaCuts& cuts)
{
	int** out_pixel_values = (int**)malloc(sizeof(int*)*cuts.height);

	for (int i = 0; i < cuts.height; i++)
	{
		out_pixel_values[i] = (int*)malloc(sizeof(int)* cuts.width);
		for (int j = 0; j < cuts.width; j++) {
			out_pixel_values[i][j] = 0;
		}
	}
	for (int i = 0; i < cuts.graph_size1; i++)
	{

		int row = i / cuts.width1, col = i % cuts.width1;

		if (row >= 0 && col >= 0 && row <= cuts.height - 1 && col <= cuts.width - 1)
			out_pixel_values[row][col] = (int)(float(cuts.pixelLabel[row*cuts.width + col])/(cuts.num_Labels-1) * 255);
	}
	FILE* fp = fopen(filename, "w");

	fprintf(fp, "%c", 'P');
	fprintf(fp, "%c", '2');
	fprintf(fp, "%c", '\n');
	fprintf(fp, "%d %c %d %c ", cuts.width, ' ', cuts.height, '\n');
	fprintf(fp, "%d %c", 255, '\n');

	for (int i = 0; i<cuts.height; i++)
	{
		for (int j = 0; j<cuts.width; j++)
		{
			fprintf(fp, "%d\n", out_pixel_values[i][j]);
		}
	}
	fclose(fp);
	for (int i = 0; i < cuts.height; i++)
		free(out_pixel_values[i]);
	free(out_pixel_values);
}

void loadFile(inputfile_t& input_file)
{
	printf("enterd\n");
	int &width = input_file.width;
	int &height = input_file.height;
	int &nLabels = input_file.numOfLabels;
	
	int *&dataCostArray = input_file.dataterm_error;
	int *&smoothCostArray = input_file.smoothness_table;
	int *&hCue = input_file.hcue;
	int *&vCue = input_file.vcue;

	char* filename = input_file.filename;
	FILE *fp = fopen(filename, "r");

	fscanf(fp, "%d %d %d", &width, &height, &nLabels);

	int i, n, x, y;
	int gt;
	for (i = 0; i < width * height; i++)
		fscanf(fp, "%d", &gt);

	dataCostArray = (int*)malloc(sizeof(int)* width * height * nLabels);

	for (int c = 0; c < nLabels; c++) {
		n = c;
		for (i = 0; i < width * height; i++) {
			fscanf(fp, "%d", &dataCostArray[n]);
			n += nLabels;
		}
	}

	hCue = (int*)malloc(sizeof(int)* width * height);
	vCue = (int*)malloc(sizeof(int)* width * height);

	n = 0;
	for (y = 0; y < height; y++) {
		for (x = 0; x < width - 1; x++) {
			fscanf(fp, "%d", &hCue[n++]);
		}
		hCue[n++] = 0;
	}

	n = 0;
	for (y = 0; y < height - 1; y++) {
		for (x = 0; x < width; x++) {
			fscanf(fp, "%d", &vCue[n++]);
		}
	}
	for (x = 0; x < width; x++) {
		vCue[n++] = 0;
	}

	fclose(fp);
	smoothCostArray = (int*)malloc(sizeof(int)*nLabels * nLabels);
	for(int i = 0; i < nLabels; i++)
	{
		for(int j = 0; j < nLabels; j++)
		{
			smoothCostArray[i*nLabels + j] = abs(i-j);
			//smoothCostArray[i*nLabels + j] = (i == j) ? 0 : 255;
		}
	}
}