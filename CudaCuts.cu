#include "hip/hip_runtime.h"
/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       **
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **
*                                                                                          **
* Copyright (c) 2008 International Institute of Information Technology.                    **
* All rights reserved.                                                                     **
*                                                                                          **
* Permission to use, copy, modify and distribute this software and its documentation for   **
* educational purpose is hereby granted without fee, provided that the above copyright     **
* notice and this permission notice appear in all copies of this software and that you do  **
* not sell the software.                                                                   **
*                                                                                          **
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR    **
* OTHERWISE.                                                                               **
*                                                                                          **
* Created By Vibhav Vineet.                                                                **
********************************************************************************************/

#include "CudaCuts.h"

CudaCuts::CudaCuts(int width, int height, int numOfLabels, int* dataTerm_error, int* smoothness_table, int* hcue, int* vcue)
{
	int initCheck = cudaCutsInit(width, height, numOfLabels);

	dataTerm = dataTerm_error;
	smoothTerm = smoothness_table;
	hCue = hcue;
	vCue = vcue;

	printf("Compute Capability %d\n", initCheck);

	if (initCheck > 0)
	{
		printf("The grid is initialized successfully\n");
	}
	else
	if (initCheck == -1)
	{
		printf("Error: Please check the device present on the system\n");
	}

	int dataCheck = cudaCutsSetupDataTerm();

	if (dataCheck == 0)
	{
		printf("The dataterm is set properly\n");

	}
	else
	if (dataCheck == -1)
	{
		printf("Error: Please check the device present on the system\n");
	}

	int smoothCheck = cudaCutsSetupSmoothTerm();

	if (smoothCheck == 0)
	{
		printf("The smoothnessterm is set properly\n");
	}
	else
	if (smoothCheck == -1)
	{
		printf("Error: Please check the device present on the system\n");
	}


	int hcueCheck = cudaCutsSetupHCue();

	if (hcueCheck == 0)
	{
		printf("The HCue is set properly\n");
	}
	else
	if (hcueCheck == -1)
	{
		printf("Error: Please check the device present on the system\n");
	}

	int vcueCheck = cudaCutsSetupVCue();


	if (vcueCheck == 0)
	{
		printf("The VCue is set properly\n");
	}
	else
	if (vcueCheck == -1)
	{
		printf("Error: Please check the device present on the system\n");
	}
}

CudaCuts::~CudaCuts()
{
	cudaCutsFreeMem();
}

void CudaCuts::run(std::vector<int> labels)
{
	for(int i = 0; i < num_Labels; i++)
	//for(int i = num_Labels-1; i >= 0 ; i--)
	{
		cudaCutsResetMem();
		cudaCutsSetupAlpha(labels[i]);
		cudaCutsSetupGraph();
		cudaCutsStochasticOptimize();
	}
	cudaCutsGetResult();
}

/********************************************************************
* cudaCutsInit(width, height, numOfLabels) function sets the      **
* width, height and numOfLabels of grid. It also initializes the  **
* block size  on the device and finds the total number of blocks  **
* running in parallel on the device. It calls checkDevice         **
* function which checks whether CUDA compatible device is present **
* on the system or not. It allocates the memory on the host and   **
* the device for the arrays which are required through the        **
* function call h_mem_init and segment_init respectively. This    **
* function returns 0 on success or -1 on failure if there is no   **
* * * CUDA compatible device is present on the system             **
* *****************************************************************/

int CudaCuts::cudaCutsInit(int widthGrid, int heightGrid, int labels)
{
	deviceCount = checkDevice();

	printf("No. of devices %d\n", deviceCount);
	if (deviceCount < 1)
		return -1;

	int cuda_device = 0;

	hipSetDevice(cuda_device);

	hipDeviceProp_t device_properties;

	CUDA_SAFE_CALL(hipGetDeviceProperties(&device_properties, cuda_device));

	if ((3 <= device_properties.major) && (device_properties.minor < 1))
		deviceCheck = 2;
	else
	if ((3 <= device_properties.major) && (device_properties.minor >= 1))
		deviceCheck = 1;
	else
		deviceCheck = 0;



	width = widthGrid;
	height = heightGrid;
	num_Labels = labels;

	blocks_x = 1;
	blocks_y = 1;
	num_of_blocks = 1;

	num_of_threads_per_block = 256;
	threads_x = 32;
	threads_y = 8;

	width1 = threads_x * ((int)ceil((float)width / (float)threads_x));
	height1 = threads_y * ((int)ceil((float)height / (float)threads_y));

	graph_size = width * height;
	graph_size1 = width1 * height1;
	size_int = sizeof(int)* graph_size1;

	blocks_x = (int)((ceil)((float)width1 / (float)threads_x));
	blocks_y = (int)((ceil)((float)height1 / (float)threads_y));

	num_of_blocks = (int)((ceil)((float)graph_size1 / (float)num_of_threads_per_block));

	h_mem_init();
	d_mem_init();
	cueValues = 0;

	return deviceCheck;

}


int CudaCuts::checkDevice()
{
	int deviceCount;

	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		return -1;
	}


	return deviceCount;
}


void CudaCuts::h_mem_init()
{
	h_reset_mem = (int*)malloc(sizeof(int)* graph_size1);
	h_graph_height = (int*)malloc(size_int);
	pixelLabel = (int*)malloc(size_int);
	h_pixel_mask = (bool*)malloc(sizeof(bool)* graph_size1);

	for (int i = 0; i < graph_size1; i++)
	{
		pixelLabel[i] = 0;
		h_graph_height[i] = 0;
	}

	for (int i = 0; i < graph_size1; i++)
	{
		h_reset_mem[i] = 0;
	}
}


void CudaCuts::d_mem_init()
{
	CUDA_SAFE_CALL(hipMalloc((void**)&d_left_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_right_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_down_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_up_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_push_reser, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_sink_weight, sizeof(int)* graph_size1));

	CUDA_SAFE_CALL(hipMalloc((void**)&s_left_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&s_right_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&s_down_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&s_up_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&s_push_reser, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&s_sink_weight, sizeof(int)* graph_size1));


	CUDA_SAFE_CALL(hipMalloc((void**)&d_stochastic, sizeof(int)* num_of_blocks));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_stochastic_pixel, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_terminate, sizeof(int)* num_of_blocks));


	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );
	//CUDA_SAFE_CALL( hipMalloc((void**)&d_sink_weight, sizeof(int) * graph_size1 ) );


	CUDA_SAFE_CALL(hipMalloc((void**)&d_pull_left, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_pull_right, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_pull_down, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_pull_up, sizeof(int)* graph_size1));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_graph_heightr, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_graph_heightw, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_relabel_mask, sizeof(int)* graph_size1));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_pixel_mask, sizeof(bool)*graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_over, sizeof(bool)* 1));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_counter, sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void **)&dPixelLabel, sizeof(int)* width1 * height1));
	CUDA_SAFE_CALL(hipMemcpy(d_left_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_right_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_down_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_up_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_push_reser, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_sink_weight, h_reset_mem, sizeof(int)* graph_size1, hipMemcpyHostToDevice));

	h_relabel_mask = (int*)malloc(sizeof(int)*width1*height1);

	h_stochastic = (int *)malloc(sizeof(int)* num_of_blocks);
	h_stochastic_pixel = (int *)malloc(sizeof(int)* graph_size1);



	for (int i = 0; i < graph_size1; i++)
		h_relabel_mask[i] = 1;


	CUDA_SAFE_CALL(hipMemcpy(d_relabel_mask, h_relabel_mask, sizeof(int)* graph_size1, hipMemcpyHostToDevice));

	int *dpixlab = (int*)malloc(sizeof(int)*width1*height1);

	for (int i = 0; i < width1 * height1; i++)
	{
		dpixlab[i] = 0;
		h_stochastic_pixel[i] = 1;
	}

	for (int i = 0; i < num_of_blocks; i++)
	{
		h_stochastic[i] = 1;
	}

	CUDA_SAFE_CALL(hipMemcpy(d_stochastic, h_stochastic, sizeof(int)* num_of_blocks, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_terminate, h_stochastic, sizeof(int)* num_of_blocks, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_stochastic_pixel, h_stochastic_pixel, sizeof(int)* graph_size1, hipMemcpyHostToDevice));


	CUDA_SAFE_CALL(hipMemcpy(dPixelLabel, dpixlab, sizeof(int)* width1 * height1, hipMemcpyHostToDevice));

	free(dpixlab);
}


int CudaCuts::cudaCutsSetupDataTerm()
{
	if (deviceCheck < 1)
		return -1;

	CUDA_SAFE_CALL(hipMalloc((void **)&dDataTerm, sizeof(int)* width * height * num_Labels));

	CUDA_SAFE_CALL(hipMemcpy(dDataTerm, dataTerm, sizeof(int)* width * height * num_Labels, hipMemcpyHostToDevice));

	return 0;
}


int CudaCuts::cudaCutsSetupSmoothTerm()
{
	if (deviceCheck < 1)
		return -1;

	CUDA_SAFE_CALL(hipMalloc((void **)&dSmoothTerm, sizeof(int)* num_Labels * num_Labels));

	CUDA_SAFE_CALL(hipMemcpy(dSmoothTerm, smoothTerm, sizeof(int)* num_Labels * num_Labels, hipMemcpyHostToDevice));

	return 0;
}

int CudaCuts::cudaCutsSetupHCue()
{

	if (deviceCheck < 1)
		return -1;

	CUDA_SAFE_CALL(hipMalloc((void **)&dHcue, sizeof(int)* width * height));

	CUDA_SAFE_CALL(hipMemcpy(dHcue, hCue, sizeof(int)* width * height, hipMemcpyHostToDevice));

	cueValues = 1;

	return 0;
}

int CudaCuts::cudaCutsSetupVCue()
{
	if (deviceCheck < 1)
		return -1;

	CUDA_SAFE_CALL(hipMalloc((void **)&dVcue, sizeof(int)* width * height));

	CUDA_SAFE_CALL(hipMemcpy(dVcue, vCue, sizeof(int)* width * height, hipMemcpyHostToDevice));

	return 0;
}

int CudaCuts::cudaCutsSetupAlpha(int alpha)
{
	alpha_label = alpha;
	return alpha_label;
}

int CudaCuts::cudaCutsSetupGraph()
{

	if (deviceCheck < 1)
		return -1;

	for (int i = 0; i < graph_size1; i++)
	{
		h_reset_mem[i] = 0;
		h_graph_height[i] = 0;
	}

	int blockEdge = (int)((ceil)((float)(width * height) / (float)256));
	dim3 block_weight(256, 1, 1);
	dim3 grid_weight(blockEdge, 1, 1);

	if (cueValues == 1)
	{
		CudaWeightCue << < grid_weight, block_weight >> >(alpha_label, d_left_weight, d_right_weight, d_down_weight,
			d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm,
			dSmoothTerm, dHcue, dVcue, width, height, num_Labels);
	}
	else
	{
		CudaWeight << < grid_weight, block_weight >> >(alpha_label, d_left_weight, d_right_weight, d_down_weight,
			d_up_weight, d_push_reser, d_sink_weight, dPixelLabel, dDataTerm,
			dSmoothTerm, width, height, num_Labels);

	}

	int *temp_left_weight, *temp_right_weight, *temp_down_weight, *temp_up_weight, *temp_source_weight, *temp_terminal_weight;

	CUDA_SAFE_CALL(hipMalloc((void **)&temp_left_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void **)&temp_right_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void **)&temp_down_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void **)&temp_up_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void **)&temp_source_weight, sizeof(int)* graph_size1));
	CUDA_SAFE_CALL(hipMalloc((void **)&temp_terminal_weight, sizeof(int)* graph_size1));

	int blockEdge1 = (int)((ceil)((float)(width1 * height1) / (float)256));
	dim3 block_weight1(256, 1, 1);
	dim3 grid_weight1(blockEdge1, 1, 1);

	adjustedgeweight << <grid_weight1, block_weight1 >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser,
		d_sink_weight, temp_left_weight, temp_right_weight, temp_down_weight, temp_up_weight,
		temp_source_weight, temp_terminal_weight, width, height, graph_size, width1,
		height1, graph_size1);

	copyedgeweight << <grid_weight1, block_weight1 >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_push_reser, d_sink_weight,
		temp_left_weight, temp_right_weight, temp_down_weight, temp_up_weight, temp_source_weight,
		temp_terminal_weight, d_pull_left, d_pull_right, d_pull_down, d_pull_up, d_relabel_mask,
		d_graph_heightr, d_graph_heightw, width, height, graph_size, width1, height1, graph_size1);

	CUDA_SAFE_CALL(hipFree(temp_left_weight));
	CUDA_SAFE_CALL(hipFree(temp_right_weight));
	CUDA_SAFE_CALL(hipFree(temp_up_weight));
	CUDA_SAFE_CALL(hipFree(temp_down_weight));
	CUDA_SAFE_CALL(hipFree(temp_source_weight));
	CUDA_SAFE_CALL(hipFree(temp_terminal_weight));
	return 0;
}

int CudaCuts::cudaCutsAtomicOptimize()
{
	if (deviceCheck < 1)
	{
		return -1;
	}

	cudaCutsAtomic();

	bfsLabeling();

	return 0;

}

int CudaCuts::cudaCutsStochasticOptimize()
{
	if (deviceCheck < 1)
	{
		return -1;
	}

	cudaCutsStochastic();

	bfsLabeling();

	return 0;

}

void CudaCuts::cudaCutsAtomic()
{

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block, 1, 1);
	dim3 d_grid(num_of_blocks, 1, 1);

	bool finish = true;

	counter = num_of_blocks;

	int numThreadsEnd = 256, numBlocksEnd = 1;
	if (numThreadsEnd > counter)
	{
		numBlocksEnd = 1;
		numThreadsEnd = counter;
	}
	else
	{
		numBlocksEnd = (int)ceil(counter / (double)numThreadsEnd);
	}

	dim3 End_block(numThreadsEnd, 1, 1);
	dim3 End_grid(numBlocksEnd, 1, 1);

	int *d_counter;

	bool *d_finish;
	for (int i = 0; i < num_of_blocks; i++)
	{
		h_stochastic[i] = 0;
	}

	CUDA_SAFE_CALL(hipMalloc((void**)&d_counter, sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_finish, sizeof(bool)));

	CUDA_SAFE_CALL(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	counter = 0;
	int *d_relabel;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_relabel, sizeof(int)));

	int h_relabel = 0;

	int block_num = width1 / 32;

	int *d_block_num;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_block_num, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));

	int h_count_blocks = num_of_blocks;
	int *d_count_blocks;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_count_blocks, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));

	h_count_blocks = 0;



	CUDA_SAFE_CALL(hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));

	counter = 1;
	kernel_push1_start_atomic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
		d_sink_weight, d_push_reser,
		d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
		graph_size1, width1, height1, d_relabel, d_stochastic, d_counter, d_finish);

	int h_terminate_condition = 0;
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	do
	{

		if (counter % 10 == 0)
		{
			finish = true;
			CUDA_SAFE_CALL(hipMemcpy(d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1 << <grid_push, block_push >> >(d_push_reser, s_push_reser, d_count_blocks, d_finish, d_block_num, width1);
			CUDA_SAFE_CALL(hipMemcpy(&finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
			if (finish == false)
				h_terminate_condition++;
		}
		if (counter % 11 == 0)
		{
			CUDA_SAFE_CALL(hipMemset(d_terminate, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0;
			CUDA_SAFE_CALL(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_atomic2 << <grid_push, block_push >> >(d_terminate, d_push_reser, s_push_reser, d_block_num, width1);

			kernel_End << <End_grid, End_block >> >(d_terminate, d_count_blocks, d_counter);

		}

		if (counter % 2 == 0)
		{

			kernel_push1_atomic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1);

			/*kernel_push2_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
			d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up,
			d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height,
			graph_size1, width1 , height1 );
			*/
			kernel_relabel_atomic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1);
		}
		else
		{
			kernel_push1_atomic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1);

			/*kernel_push2_atomic<<<grid_push,block_push>>>(d_left_weight,d_right_weight, d_down_weight, d_up_weight,
			d_sink_weight, d_push_reser,d_pull_left, d_pull_right, d_pull_down, d_pull_up,
			d_relabel_mask,d_graph_heightr,d_graph_heightw, graph_size,width,height,
			graph_size1, width1 , height1);
			*/
			kernel_relabel_atomic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1);

		}
		counter++;
	} while (h_terminate_condition != 2);

	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));
	float time;
	CUDA_SAFE_CALL(hipEventElapsedTime(&time, start, stop));
	printf("TT Cuts :: %f ms\n", time);

}


void CudaCuts::cudaCutsStochastic()
{

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block, 1, 1);
	dim3 d_grid(num_of_blocks, 1, 1);

	bool finish = true;

	counter = num_of_blocks;

	int numThreadsEnd = 256, numBlocksEnd = 1;
	if (numThreadsEnd > counter)
	{
		numBlocksEnd = 1;
		numThreadsEnd = counter;
	}
	else
	{
		numBlocksEnd = (int)ceil(counter / (double)numThreadsEnd);
	}

	dim3 End_block(numThreadsEnd, 1, 1);
	dim3 End_grid(numBlocksEnd, 1, 1);




	bool *d_finish;
	for (int i = 0; i < num_of_blocks; i++)
	{
		h_stochastic[i] = 0;
	}

	CUDA_SAFE_CALL(hipMalloc((void**)&d_counter, sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_finish, sizeof(bool)));

	CUDA_SAFE_CALL(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	counter = 0;
	int *d_relabel;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_relabel, sizeof(int)));

	int h_relabel = 0;


	int block_num = width1 / 32;

	int *d_block_num;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_block_num, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_block_num, &block_num, sizeof(int), hipMemcpyHostToDevice));


	int h_count_blocks = num_of_blocks;
	int *d_count_blocks;

	CUDA_SAFE_CALL(hipMalloc((void**)&d_count_blocks, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));

	h_count_blocks = 0;

	CUDA_SAFE_CALL(hipMemcpy(d_relabel, &h_relabel, sizeof(int), hipMemcpyHostToDevice));

	counter = 1;
	kernel_push1_start_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
		d_sink_weight, d_push_reser,
		d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
		graph_size1, width1, height1, d_relabel, d_stochastic, d_counter, d_finish);
	int h_terminate_condition = 0;
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//for (int i = 0 ; i < 400; i++ )
	do
	{
		if (counter % 10 == 0)
		{
			finish = true;
			CUDA_SAFE_CALL(hipMemcpy(d_finish, &finish, sizeof(bool), hipMemcpyHostToDevice));
			kernel_push_stochastic1 << <grid_push, block_push >> >(d_push_reser, s_push_reser, d_count_blocks, d_finish, d_block_num, width1);
			CUDA_SAFE_CALL(hipMemcpy(&finish, d_finish, sizeof(bool), hipMemcpyDeviceToHost));
		}
		if (counter % 11 == 0)
		{
			CUDA_SAFE_CALL(hipMemset(d_stochastic, 0, sizeof(int)*num_of_blocks));
			CUDA_SAFE_CALL(hipMemset(d_terminate, 0, sizeof(int)*num_of_blocks));
			h_count_blocks = 0;
			CUDA_SAFE_CALL(hipMemcpy(d_count_blocks, &h_count_blocks, sizeof(int), hipMemcpyHostToDevice));
			kernel_push_stochastic2 << <grid_push, block_push >> >(d_terminate, d_relabel_mask, d_push_reser, s_push_reser, d_stochastic, d_block_num, width1);

			kernel_End << <End_grid, End_block >> >(d_terminate, d_count_blocks, d_counter);

			//if ( finish == false ) printf("%d \n",counter);
			if (finish == false && counter % 121 != 0 && counter > 0)
				h_terminate_condition++;

		}
		if (counter % 2 == 0)
		{

			kernel_push1_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

			/*kernel_push2_stochastic<<<grid_push, block_push>>>( d_left_weight, d_right_weight, d_down_weight, d_up_weight,
			d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
			d_relabel_mask, d_graph_heightr, d_graph_heightw,
			graph_size, width, height, graph_size1, width1, height1, d_relabel, d_stochastic, d_counter, d_finish, d_block_num) ;
			*/
			kernel_relabel_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,/*d_pull_left, d_pull_right, d_pull_down, d_pull_up,*/
				d_relabel_mask, d_graph_heightr, d_graph_heightw, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

		}
		else
		{
			kernel_push1_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);


			/*kernel_push2_stochastic<<<grid_push, block_push>>>( d_left_weight, d_right_weight, d_down_weight, d_up_weight,
			d_sink_weight, d_push_reser, d_pull_left, d_pull_right, d_pull_down, d_pull_up,
			d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height, graph_size1,
			width1, height1, d_relabel, d_stochastic, d_counter, d_finish, d_block_num) ;
			*/

			kernel_relabel_stochastic << <grid_push, block_push >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight,
				d_sink_weight, d_push_reser,
				d_relabel_mask, d_graph_heightw, d_graph_heightr, graph_size, width, height,
				graph_size1, width1, height1, d_stochastic, d_block_num);

		}
		counter++;
	} while (h_terminate_condition == 0 && counter < 500);


	CUDA_SAFE_CALL(hipEventRecord(stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize(stop));
	float time;
	CUDA_SAFE_CALL(hipEventElapsedTime(&time, start, stop));
	printf("TT Cuts :: %f ms\n", time);

}

void CudaCuts::bfsLabeling()
{

	dim3 block_push(threads_x, threads_y, 1);
	dim3 grid_push(blocks_x, blocks_y, 1);

	dim3 d_block(num_of_threads_per_block, 1, 1);
	dim3 d_grid(num_of_blocks, 1, 1);

	CUDA_SAFE_CALL(hipMemcpy(d_graph_heightr, h_graph_height, size_int, hipMemcpyHostToDevice));

	for (int i = 0; i < graph_size; i++)
		h_pixel_mask[i] = true;

	CUDA_SAFE_CALL(hipMemcpy(d_pixel_mask, h_pixel_mask, sizeof(bool)* graph_size1, hipMemcpyHostToDevice));

	kernel_bfs_t << <d_grid, d_block, 0 >> >(d_push_reser, d_sink_weight, d_graph_heightr, d_pixel_mask, graph_size, width, height, graph_size1, width1, height1);
	counter = 1;

	CUDA_SAFE_CALL(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));

	do
	{
		h_over = false;

		CUDA_SAFE_CALL(hipMemcpy(d_over, &h_over, sizeof(bool), hipMemcpyHostToDevice));

		kernel_bfs << < d_grid, d_block, 0 >> >(d_left_weight, d_right_weight, d_down_weight, d_up_weight, d_graph_heightr, d_pixel_mask,
			graph_size, width, height, graph_size1, width1, height1, d_over, d_counter);

		CUDA_SAFE_CALL(hipMemcpy(&h_over, d_over, sizeof(bool), hipMemcpyDeviceToHost));

		counter++;

		CUDA_SAFE_CALL(hipMemcpy(d_counter, &counter, sizeof(int), hipMemcpyHostToDevice));
	} while (h_over);

	updatePixelLabel<<<d_grid, d_block, 0>>>(alpha_label, dPixelLabel, d_graph_heightr, graph_size1, width, height, width1, height1);
}


int CudaCuts::cudaCutsGetResult()
{
	if (deviceCheck < 1)
		return -1;

	CUDA_SAFE_CALL(hipMemcpy(pixelLabel, dPixelLabel, size_int, hipMemcpyDeviceToHost));

	return 0;

}

void CudaCuts::cudaCutsFreeMem()
{
	free(h_reset_mem);
	free(h_graph_height);
	free(pixelLabel);
	free(h_pixel_mask);
	
	free(h_relabel_mask);
	free(h_stochastic);
	free(h_stochastic_pixel);
	
	free(hCue);
	free(vCue);
	free(dataTerm);
	free(smoothTerm);

	CUDA_SAFE_CALL(hipFree(d_left_weight));
	CUDA_SAFE_CALL(hipFree(d_right_weight));
	CUDA_SAFE_CALL(hipFree(d_down_weight));
	CUDA_SAFE_CALL(hipFree(d_up_weight));
	CUDA_SAFE_CALL(hipFree(d_sink_weight));
	CUDA_SAFE_CALL(hipFree(d_push_reser));

	CUDA_SAFE_CALL(hipFree(d_pull_left));
	CUDA_SAFE_CALL(hipFree(d_pull_right));
	CUDA_SAFE_CALL(hipFree(d_pull_down));
	CUDA_SAFE_CALL(hipFree(d_pull_up));

	CUDA_SAFE_CALL(hipFree(d_graph_heightr));
	CUDA_SAFE_CALL(hipFree(d_graph_heightw));

	CUDA_SAFE_CALL(hipFree(s_left_weight));
	CUDA_SAFE_CALL(hipFree(s_right_weight));
	CUDA_SAFE_CALL(hipFree(s_down_weight));
	CUDA_SAFE_CALL(hipFree(s_up_weight));
	CUDA_SAFE_CALL(hipFree(s_push_reser));
	CUDA_SAFE_CALL(hipFree(s_sink_weight));
	
	
	CUDA_SAFE_CALL(hipFree(d_stochastic));
	CUDA_SAFE_CALL(hipFree(d_stochastic_pixel));
	CUDA_SAFE_CALL(hipFree(d_terminate));
	
	CUDA_SAFE_CALL(hipFree(d_relabel_mask));
	
	CUDA_SAFE_CALL(hipFree(d_pixel_mask));
	CUDA_SAFE_CALL(hipFree(d_over));
	CUDA_SAFE_CALL(hipFree(d_counter));
	
	CUDA_SAFE_CALL(hipFree(dPixelLabel));
}

void CudaCuts::cudaCutsResetMem()
{
	hipMemset(d_left_weight, 0, sizeof(int)* graph_size1);
	hipMemset(d_right_weight, 0, sizeof(int)* graph_size1);
	hipMemset(d_down_weight, 0, sizeof(int)* graph_size1);
	hipMemset(d_up_weight, 0, sizeof(int)* graph_size1);
	hipMemset(d_push_reser, 0, sizeof(int)* graph_size1);
	hipMemset(d_sink_weight, 0, sizeof(int)* graph_size1);
}
